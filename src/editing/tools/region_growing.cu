#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/editing/tools/region_growing.h>
#include <neural-graphics-primitives/editing/tools/selection_utils.h>
#include <neural-graphics-primitives/common_nerf.h>
#include <cmath>
#include <tiny-cuda-nn/common_device.h>
#include <functional>

NGP_NAMESPACE_BEGIN

//GUI Button "Clear Selection"
// Launched by GrowingSelection::reset_growing selection grid
void RegionGrowing::reset_growing(const std::vector<uint32_t>& selected_cells, int growing_level) {
    // Copy the density grid
    std::cout << "reset_growing() " << std::endl;
    m_density_grid_host.resize(m_density_grid.size());
    m_density_grid.copy_to_host(m_density_grid_host);

    // Reset the selection grid (0 empty, 1 selected)	
    m_selection_grid_bitfield = std::vector<uint8_t>(m_density_grid_bitfield.size(), 0);

    // Reset the growing queue
    m_growing_queue = std::queue<uint32_t>();

    uint32_t n_rays = selected_cells.size();

    // Reset the points (used for visualization)
    m_selection_points.clear();
    m_selection_cell_idx.clear();
    m_selection_points.reserve(n_rays);
    m_selection_cell_idx.reserve(n_rays);

    for (int i = 0; i < n_rays; i++) {
        uint32_t cell_idx = selected_cells[i];
        uint32_t level = cell_idx / NERF_GRIDVOLUME();

        // If it's bigger than the requested level, discard it
        if (level > growing_level) {
            continue;
        }
        // If it is smaller then uplift!
        if (level < growing_level) {
            cell_idx = get_upper_cell_idx(cell_idx, growing_level);
        };
        level = cell_idx / NERF_GRIDVOLUME();

        // Add all pixels to their reprojected coordinate in the queue
        m_growing_queue.push(cell_idx);

        // Add visualization points
        // Invert morton coordinates to get xyz
        uint32_t pos_idx = cell_idx % NERF_GRIDVOLUME();
        uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
        uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
        uint32_t z = tcnn::morton3D_invert(pos_idx>>2);
        m_selection_points.push_back(get_cell_pos(x, y, z, level));
        m_selection_cell_idx.push_back(cell_idx);
    }
}

void RegionGrowing::upscale_selection(int current_level) {
    // If the current level is already the maximum cascade, we can't upscale
    if (current_level == m_max_cascade)
        return;
    // Otherwise, upscale everything
    m_growing_level = current_level + 1;

    // Reset the bitfield grid
    std::fill(m_selection_grid_bitfield.begin(), m_selection_grid_bitfield.end(), 0);
    
    // Upscale the existing points
    std::vector<uint32_t> new_cell_indices;
    m_selection_points = std::vector<Eigen::Vector3f>();
    for (const auto cell_idx: m_selection_cell_idx) {
        uint32_t new_cell_idx = get_upper_cell_idx(cell_idx, m_growing_level);
        uint32_t new_pos_idx = new_cell_idx % (NERF_GRIDVOLUME());
        uint32_t x = tcnn::morton3D_invert(new_pos_idx>>0);
        uint32_t y = tcnn::morton3D_invert(new_pos_idx>>1);
        uint32_t z = tcnn::morton3D_invert(new_pos_idx>>2);
        Eigen::Vector3f cell_pos = get_cell_pos(x, y, z, m_growing_level);
        m_selection_points.push_back(cell_pos);
        new_cell_indices.push_back(new_cell_idx);
        set_bitfield_at(new_pos_idx, m_growing_level, true, m_selection_grid_bitfield.data());
    }
    m_selection_cell_idx = new_cell_indices;

    // Upscale the growing queue too
    std::queue<uint32_t> new_growing_queue;
    while (!m_growing_queue.empty()) {
        uint32_t current_cell = m_growing_queue.front();
        m_growing_queue.pop();
        new_growing_queue.push(get_upper_cell_idx(current_cell, m_growing_level));
    }
    m_growing_queue = new_growing_queue;
}

//GUI Button "Grow region" and "Grow Far", second function
void RegionGrowing::grow_region(bool ed_flag, float density_threshold, ERegionGrowingMode region_growing_mode, int growing_level, int growing_steps) {
    // Make sure we can actually grow!
    if (m_growing_queue.empty()) {
        std::cout << "Growing queue is empty!" << std::endl;
        return;
    }
    m_growing_level = growing_level;            //attenzione a growing_level che sta a 0 

    int i = 1;

    if (region_growing_mode == ERegionGrowingMode::Manual) {
        while (!m_growing_queue.empty() && i <= growing_steps) {
            uint32_t current_cell = m_growing_queue.front();                //current_cell = m_selection_cell_idx 
            float current_density = m_density_grid_host[current_cell];      //con operatori secondari, di base m_density_grid_host è vuoto 
            m_growing_queue.pop();

            // Get position (with corresponding level) to fetch neighbours
            uint32_t level = current_cell / (NERF_GRIDVOLUME());
            uint32_t pos_idx = current_cell % (NERF_GRIDVOLUME());

            // Sample accepted only if at requested level, statisfying density threshold and not already selected!
            if (!get_bitfield_at(pos_idx, level, m_selection_grid_bitfield.data())) {                           //ERRORE Access violation reading location 0x0000000000063E23
                if(current_density >= density_threshold){                   //serve current_density + alta
                if (level == m_growing_level) {
                // Test whether the new sample touches the boundary, if yes then upscale!
                if (is_boundary(pos_idx)) {
                    std::cout << "UPSAMPLING" << std::endl;
                    upscale_selection(m_growing_level);
                    // Don´t forget to also upscale the current cell!
                    current_cell = get_upper_cell_idx(current_cell, m_growing_level);
                    level = current_cell / (NERF_GRIDVOLUME());
                    pos_idx = current_cell % (NERF_GRIDVOLUME());
                }

                // Invert morton coordinates to get xyz
                uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
                uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
                uint32_t z = tcnn::morton3D_invert(pos_idx>>2);
                // Add possible neighbours
                add_neighbours(m_growing_queue, x, y, z, level);

                // Mark the current cell
                Eigen::Vector3f cell_pos = get_cell_pos(x, y, z, level);
                m_selection_points.push_back(cell_pos);
                m_selection_cell_idx.push_back(current_cell);
                set_bitfield_at(pos_idx, level, true, m_selection_grid_bitfield.data());
                //std::cout << "m_selection_cell_idx: " << current_cell << std::endl;
            }
                }
            }
            i++;
        }
 
        //SI POTREBBE AGGIUNGERE QUI UN CONTROLLO SUI PUNTI DUPLICATI POST GROWING
		//aggiungere tutto ad un set e poi assegnare il contenuto a m_selection_points, 
        //ma bisognerebbe anche sistemare m_selection_cell_idx e m_selection_grid_bitfield

        //Si prosegue solo con il Grow Far Button
        if(ed_flag){
            equidistant_points(min_ed_points_threshold);
        }    
    }        
    // std::cout << "Selected " << m_selection_points.size() << " points overall" << std::endl;
}

bool not_zero_coordinate(Eigen::Vector3f point_to_check) {
    //std::cout << "point_to_check: " << point_to_check << std::endl;
    if (point_to_check == Eigen::Vector3f(0.0f, 0.0f, 0.0f)) {
        std::cout << " Zero Coordinate Point discarded------------------------------------------------------------------------------- " << std::endl;
        return false;
    }
    else {
        return true;
    }
}

//DA FIXARE, NON PRENDE SEMPRE PUNTI DISTANTI IN TERMINI DI COORDINATE######################################################################
//Seleziona in modo uniforme solo alcuni punti superficiali distanti; si ferma al raggiungimento della soglia minima
void RegionGrowing::equidistant_points(int min_ed_points_threshold) {
    std::cout << "PRE m_selection_points size: "<< m_selection_points.size() << std::endl;
    //Vettori temporanei 
    std::vector<Eigen::Vector3f> m_temp_points;
    std::vector<uint32_t> m_temp_idx;

    //Ogni quanti punti bisogna salvarne 1 (per prendere punti distanti in modo uniforme)
    int interval = static_cast<int>(std::round(static_cast<double>(m_selection_points.size()) /  min_ed_points_threshold));
    int count = 0;                                                                                  //counter per scorrere l'array
    if (interval == 0){
         std::cout << "RegionGrowing::equidistant_points() failed: Not enough superficial points selected. Try with a higher growing level."<< std::endl;
        return;
    }

    selection_map selection_mapObj;

    for (int i = 0; i < m_selection_points.size() && m_temp_points.size() < max_ed_points_limit; i++) {
        if ((count % interval == 0) && (not_zero_coordinate(m_selection_points[i]))) {
            m_temp_points.push_back(m_selection_points[i]);
            m_temp_idx.push_back(m_selection_cell_idx[i]);
            //Aggiornamento Mappa utilizzata dallo SPLIT Button
            selection_mapObj.add_to_privateMap(m_selection_cell_idx[i], m_selection_points[i]);        
            //vstd::cout << "Growing point added: "<< i << " with id: " << id << std::endl;
        }
        count++;
    }

    // Sostituisce i vecchi vettori con quelli aggiornati
    m_selection_points = m_temp_points;
    m_selection_cell_idx = m_temp_idx;
    std::cout << "POST m_selection_points size: "<< m_selection_points.size() << std::endl;
}  

//DA FIXARE, NON PRENDE SEMPRE PUNTI DISTANTI IN TERMINI DI COORDINATE######################################################################
//Seleziona in modo uniforme solo alcuni punti superficiali distanti; intervallo scelto dall'utente; continua finchè non supera la soglia minima
void RegionGrowing::equidistant_points(int min_ed_points_threshold, int interval) {
    std::cout << "PRE m_selection_points size: "<< m_selection_points.size() << std::endl;
    //Vettori temporanei 
    std::vector<Eigen::Vector3f> m_temp_points;
    std::vector<uint32_t> m_temp_idx;
    int count = 0;                                                                              //counter per scorrere l'array
    selection_map selection_mapObj;

    for (int i = 0; i < m_selection_points.size() && m_temp_points.size() < max_ed_points_limit; i++) {
        if (count % interval == 0) {
            m_temp_points.push_back(m_selection_points[i]);
            m_temp_idx.push_back(m_selection_cell_idx[i]);
            selection_mapObj.add_to_privateMap(m_selection_cell_idx[i], m_selection_points[i]);
            //std::cout << "Growing point added A: "<< i << std::endl;
        }
        count++;
    }
  
    int interval2 = 0;
    int remaining_ud_points = min_ed_points_threshold - m_temp_points.size();
    if ( remaining_ud_points > 0) {
        interval2 = static_cast<int>(m_selection_points.size() / remaining_ud_points);
        if (interval == 0){
            std::cout << "RegionGrowing::equidistant_points() failed: Not enough superficial points selected. Try with a higher growing level."<< std::endl;
            return;
        }
        for (int i = 0; i < m_selection_points.size() && remaining_ud_points > 0 && m_temp_points.size() < max_ed_points_limit; i++) {
            if (count % interval2 == 0) {
                auto it = std::find(m_temp_points.begin(), m_temp_points.end(), m_selection_points[i]); //restituisce puntatore a ultimo elemento, se non trova l'oggetto
                if (it == m_temp_points.end()) {                            //se l'oggetto non è presente, viene aggiunto 
                    m_temp_points.push_back(m_selection_points[i]);
                    m_temp_idx.push_back(m_selection_cell_idx[i]);
                    //INSERIMENTO NELLA MAPPA NON TESTATO###########
                    selection_mapObj.add_to_privateMap(m_selection_cell_idx[i], m_selection_points[i]); 
                    remaining_ud_points--;
                    //std::cout << "Growing point added B: "<< i << std::endl;
                }
                
            }
            count++;
        }
    }
    
    // Sostituisci i vecchi vettori con quelli aggiornati
    m_selection_points = m_temp_points;
    m_selection_cell_idx = m_temp_idx;
    std::cout << "POST m_selection_points size: "<< m_selection_points.size() << std::endl;
}      

// Queue needs to be copied because we'll exhaust it
template <typename T>
inline void to_json_queue(nlohmann::json& j, std::queue<T> queue) {
	std::vector<T> tmp_vec;
    tmp_vec.reserve(queue.size());
    while (!queue.empty()) {
        tmp_vec.push_back(queue.front());
        queue.pop();
    }
	to_json(j, tmp_vec);
}

template <typename T>
inline void from_json_queue(const nlohmann::json& j, std::queue<T>& queue) {
	std::vector<T> tmp_vec = j.get<std::vector<T>>();
	for (auto item: tmp_vec) {
		queue.push(item);
	}
}

nlohmann::json RegionGrowing::to_json() {
        nlohmann::json j;

        j["selection_grid_bitfield"] = m_selection_grid_bitfield;
        j["selection_points"] = m_selection_points;
        j["selection_cell_idx"] = m_selection_cell_idx;
        j["density_grid_host"] = m_density_grid_host;
        // TODO: support saving of queue
        // to_json_queue<uint32_t>(j["growing_queue"], m_growing_queue);

        return j;
    }

void RegionGrowing::load_json(nlohmann::json& j) {
    std::cout << "most" << std::endl;
    from_json(j["selection_grid_bitfield"], m_selection_grid_bitfield);
    from_json(j["selection_points"], m_selection_points);
    from_json(j["selection_cell_idx"], m_selection_cell_idx);
    from_json(j["density_grid_host"], m_density_grid_host);
    // TODO: support reloading of the queue
    // from_json_queue<uint32_t>(j["growing_queue"], m_growing_queue);
}

NGP_NAMESPACE_END